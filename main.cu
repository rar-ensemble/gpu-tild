#include "hip/hip_runtime.h"
#define MAIN
#include "globals.h"
#include "timing.h"
#include <vector>
#include <fstream>

void forces(void);
void update_pairstyles(void);
void calc_properties(void);
void initialize(void);
void write_lammps_traj(void);
void cuda_collect_x(void);
void cuda_collect_f(void);
void cuda_collect_rho(void);
void write_binary(void);
void write_struc_fac(void);
void write_grid_data(const char*, float*);
void write_kspace_data(const char*, complex<float>*);
void write_kspace_cudaComplex(const char*, hipfftComplex*);
__global__ void d_prepareDensity(int, float*, hipfftComplex*, int);
int print_timestep();
ofstream dout;
void unstack_like_device(int id, int* nn);

__global__ void cu_random_posits(float*, float*, int, int, hiprandState*);


__global__ void d_real2complex(float*, hipfftComplex*, int);
__global__ void d_complex2real(hipfftComplex*, float*, float*, int);
__global__ void d_make_step(hipfftComplex* , float*, float*, int*, int, int);
__global__ void d_multiplyComplex(hipfftComplex*, hipfftComplex*,
	hipfftComplex*, int);



int main(int argc, char** argv)
{
	main_t_in = int(time(0));
	init_t_in = main_t_in;
	std::vector<std::string> string_vec;
	for (int i = 0; i < argc; i++)
	{
		std::string arg = argv[i];
		string_vec.push_back(arg);
	}

	for (size_t i = 1; i < string_vec.size(); ++i) {
		if (string_vec[i] == "-in") {
			input_file = string_vec[++i];
		}
	}

	initialize();


	// Write initial positions to lammpstrj file
	cuda_collect_x();
	write_lammps_traj();

	dout.open("data.dat");
	dout << "# step Upe Ubond ";
	if (Dim == 2)
		dout << "Pxx Pyy Pxy ";
	else if (Dim == 3)
		dout << " Pxx Pyy Pzz Pxy Pxz Pyz ";
	for (int i = 0; i < Gausses.size(); i++) 
		dout << "Ugauss[" << i << "] ";
	for (int i = 0; i < Erfs.size(); i++) 
		dout << "Uerf[" << i << "] ";
	for (int i = 0; i < GaussErfs.size(); i++) 
		dout << "UGuassErf[" << i << "] ";
	
	dout << endl;
	
	forces();
	hipDeviceSynchronize();

	cuda_collect_rho();
	cuda_collect_x();
	calc_properties();
	if (grid_freq > 0) {
		print_t_in = int(time(0));
		hipDeviceSynchronize();

		for (int i = 0; i < ntypes; i++) {
			char nm[30];
			sprintf(nm, "rho%d.dat", i);
			write_grid_data(nm, Components[i].rho);
		}
			
		print_t_out = int(time(0));
		print_tot_time += print_t_out - print_t_in;
	}

	int die_flag = 0;
	die_flag = print_timestep();

	init_t_out = int(time(0));



	///////////////////////////////////////
	// BEGINNING OF MAIN SIMULATION LOOP //
	///////////////////////////////////////

	for (step = 1; step <= max_steps; step++) {
		
		for (int i = 0; i < n_integrators; i++)
			Integrators[i].Integrate_1();
		check_cudaError("Integrator step 1");


		forces();


		for (int i = 0; i < n_integrators; i++)
		  Integrators[i].Integrate_2();
	
		check_cudaError("Integrator step 2");


		
		// Run computes
		for (int i = 0; i < n_computes; i++) {
			if (step > Computes[i].compute_wait && step % Computes[i].compute_freq == 0) {
				
				//cout << "entering compute " << i;
				Computes[i].doCompute();
				//cout << " done!" << endl;
				check_cudaError("Compute");
			}
		}

		// I/O blocks //
		if (traj_freq > 0 && step % traj_freq == 0) {
			print_t_in = int(time(0));
			hipDeviceSynchronize();

			cuda_collect_x();
			write_lammps_traj();
			print_t_out = int(time(0));
			print_tot_time += print_t_out - print_t_in;
		}

		if (grid_freq > 0 && step % grid_freq == 0) {
			print_t_in = int(time(0));
			hipDeviceSynchronize();

			cuda_collect_rho();
			for (int i = 0; i < ntypes; i++) {
				char nm[30];
				sprintf(nm, "rho%d.dat", i);
				write_grid_data(nm, Components[i].rho);
			}

			print_t_out = int(time(0));
			print_tot_time += print_t_out - print_t_in;
		}

		if (bin_freq > 0 && step % bin_freq == 0) {
			print_t_in = int(time(0));
			hipDeviceSynchronize();

			cuda_collect_rho();
			cuda_collect_x();

			write_binary();
			print_t_out = int(time(0));
			print_tot_time += print_t_out - print_t_in;
		}


		// Write to log file, write compute results
		if (step % log_freq == 0) {
			print_t_in = int(time(0));
			hipDeviceSynchronize();

			calc_properties();

			die_flag = print_timestep();

			for (int i = 0; i < n_computes; i++)
				if (step > Computes[i].compute_wait)
					Computes[i].writeResults(i);

			print_t_out = int(time(0));
			print_tot_time += print_t_out - print_t_in;

			if (die_flag) {
				break;
			}


		}
		
	

		// Finalize time step //
		update_pairstyles();

	}// main loop over steps



	// Write resume frame and finish //
	cuda_collect_x();
	write_lammps_traj();

	main_t_out = int(time(0));
	int dt = main_t_out - main_t_in;
	cout << "Total run time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;
	
	dt = init_t_out - init_t_in;
	cout << "Total init time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;
	
	dt = bond_tot_time;
	cout << "Bond E, P props on host: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

	dt = print_tot_time;
	cout << "I/O + Comm time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

	dt = compute_tot_time;
	cout << "Computes time: " << dt / 60 << "m" << dt % 60 << "sec" << endl;

	return 0;

}


int print_timestep() {
	int die_flag = 0;
	cout << "Step " << step << " of " << max_steps << " ";

	if (do_charges == 1) {
		cout << " Electrostatic Energy: " << *electrostatic_energy;
	}

	cout << " U/V: " << Upe / V << \
		" Ubond: " << Ubond << \
		" Pdiags: " << Ptens[0] << " " << Ptens[1] << " ";

	if (Dim == 3)
		cout << Ptens[2] << " ";

	dout << step << " " << Upe << " " << Ubond << " ";
	for (int i = 0; i < n_P_comps; i++)
		dout << Ptens[i] << " ";

	
	if (!Gausses.empty()) {
		cout << "Ugauss: ";
			for (Gaussian& Iter : Gausses){
			cout << Iter.energy << " ";
			dout << Iter.energy  << " ";
			if (std::isnan(Iter.energy))
				die_flag = 1;
		}
	}

	if (!Erfs.empty()) {
	cout << "Uerf: ";
		for (Erf& Iter: Erfs) {
			cout << Iter.energy << " ";
			dout << Iter.energy << " ";
			if (std::isnan(Iter.energy))
				die_flag = 1;
		}
	}

	if (!GaussErfs.empty()) {
		cout << "UGaussErf: ";
		for (GaussianErf& Iter : GaussErfs) {
			cout << Iter.energy << " ";
			dout << Iter.energy << " ";
			if (std::isnan(Iter.energy))
				die_flag = 1;
		}
	}

	if (!Fields.empty()) {
		cout << "UFieldPhase: ";
		for (FieldPhase& Iter : Fields) {
			cout << Iter.energy << " ";
			dout << Iter.energy << " ";
			if (std::isnan(Iter.energy))
				die_flag = 1;
		}
	}

    dout << endl;
	cout<<endl;
	return die_flag;
}
