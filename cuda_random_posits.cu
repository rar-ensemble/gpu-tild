
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

// M threads per block
// Block is local in memory?
// NThreads is number of opeerations in the loop each block does
__global__ void cu_random_posits(float *x, float *L, 
	int size, int Dim, hiprandState *d_states) {

	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	hiprandState l_state; 
	l_state = d_states[ind];

	if (ind < size)
		for (int j = 0; j < Dim; j++)
			x[ind * Dim + j] = hiprand_uniform(&l_state) * L[j];
	
	d_states[ind] = l_state;
}


__global__ void init_dev_rng(unsigned int seed, hiprandState* d_states, int ns) {

	int idx = blockIdx.x * blockDim.x + threadIdx.x;//check index for >= ns

	if (idx >= ns)//this probably will not compile
		return;

	hiprand_init(seed, idx, 0, &d_states[idx]);

}