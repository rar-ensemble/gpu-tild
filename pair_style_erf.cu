#include "hip/hip_runtime.h"
#include "globals.h"
#include "pair_style_erf.h"
#include "device_utils.cuh"

__global__ void init_device_erf(float*,	float, float, float, 
	const float*, const float*, const float*, 
	const int, const int*, const int);
__global__ void init_device_erf_kspace(hipfftComplex*, hipfftComplex*, float, float, float, 
	const float*, const int, const int*, const int);
__global__ void d_real2complex(float*, hipfftComplex*, int);
__global__ void d_kSpaceMakeForce(hipfftComplex*, hipfftComplex*,
	const float*, const int*, const int, const int);
__global__ void d_extractForceComp(hipfftComplex*, hipfftComplex*, int, int, int);
__global__ void d_insertForceCompC2R(float*, hipfftComplex*, const int,
	const int, const int);
__global__ void d_divideByDimension(hipfftComplex*, int);

__global__ void d_complex2real(hipfftComplex*, float*, int);

void Erf::Initialize() {
    Initialize_Erf(initial_prefactor, sigma_squared, Rp, M, type1, type2);
}


void Erf::Initialize_Erf(float Ao, float sigma2, float Rp, 
    int alloc_size, int typ_A, int typ_P) {
	cout << "Setting up ERF pairstyle...";
	fflush(stdout);

    Initialize_PairStyle(alloc_size, typ_A, typ_P);

	init_device_erf_kspace<<<M_Grid,M_Block>>>(this->d_u_k, this->d_f_k, Ao, sigma2, Rp,
		d_L,M, d_Nx, Dim);
	hipfftExecC2C(fftplan, this->d_u_k, d_cpx1, HIPFFT_BACKWARD);
	d_complex2real << <M_Grid, M_Block >> > (d_cpx1, this->d_u, M);

	for (int j = 0; j < Dim; j++) {
		d_extractForceComp << <M_Grid, M_Block >> > (d_cpx1, this->d_f_k, j, Dim, M);
		hipfftExecC2C(fftplan, d_cpx1, d_cpx1, HIPFFT_BACKWARD);
		d_insertForceCompC2R << <M_Grid, M_Block >> > (this->d_f, d_cpx1, j, Dim, M);
	}

	// Define the potential and the force in k-space

	float k2, kv[3], k;
	float temp;
	

	for (int i = 0; i < alloc_size; i++) {
		k2 = get_k(i, kv, Dim);
		k = sqrt(k2);

		if (k2 == 0) {
			this->u_k[i] = Ao *// prefactor
				// exp(-k2 * sigma2 * 0.5f) * //Gaussian contribution = 1
				PI4 * Rp * Rp * Rp / 3 *   // step function contribution for 1
				PI4 * Rp * Rp * Rp / 3;   // step function contribution for 2
		}
		else
		{
			//FFT of step function 
			temp = PI4 * (sin(Rp * k) - Rp * k * cos(Rp * k)) / (k2 * k);

			this->u_k[i] = Ao *// prefactor
				exp(-k2 * sigma2 * 0.5f) * //Gaussian contribution of both
				temp * // step function for 1
				temp; // step function for the other
		}

		for (int j = 0; j < Dim; j++) {
			this->f_k[j][i] = -I * kv[j] * this->u_k[i];
		}

	}

    InitializeVirial();
	cout << "Done!" << endl;
}


Erf::Erf() {

}

Erf::~Erf() {

}


/* This code defines the erf function on the device,
which will then be Fourier transformed to get the force.
*/
__global__ void init_device_erf(float* u,
	float Ao, float Rp, float xi,
	const float* dL, const float* dLh, const float* ddx,
	const int dM, const int* dNx, const int dDim) {

	const int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind >= dM)
		return;

	float ro[3], ri[3], dr[3];
	for (int j = 0; j < dDim; j++)
		ro[j] = 0.f;

	d_get_r(ind, ri, dNx, ddx, dDim);

	float mdr2 = d_pbc_mdr2(ro, ri, dr, dL, dLh, dDim);
	float mdr = sqrtf(mdr2);

	float arg = (mdr - Rp) / xi;
	u[ind] = Ao * (1.0f - erff(arg));
}


/* This code defines the convolved erf function on the device in Fourier space.
 * We have fourier the Gaussian and step function contributions for each erf func.
 * The definition of xi is sqrt(2) sigma based off the comparison of 
 * 1D definition of convolv of box with gauss to get erfc and 
 * the definition we used in our papers
*/
__global__ void init_device_erf_kspace(hipfftComplex* uk,
	hipfftComplex* fk, float Ao, float sigma2, float Rp,
	const float* dL, const int dM, const int* dNx, const int dDim) {

	const int ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (ind >= dM)
		return;

	float k2, kv[3], k;

    k2 = d_get_k(ind, kv, dL, dNx, dDim);
	k = sqrt(k2);
	
	if (k2 == 0) {
		uk[ind].x = Ao *				// prefactor
			//exp(-k2 * sigma_squared * 0.5f )* //Gaussian contribution = 1
			PI4 * Rp * Rp * Rp / 3*   // step function contribution for 1
			PI4 * Rp * Rp * Rp / 3;   // step function contribution for 2
	}
	else
	{
		//FFT of step function 
		float temp = PI4 * (sin(Rp * k) - Rp * k * cos(Rp * k)) / (k2 * k);

		uk[ind].x = Ao *				//prefactor
			exp(-k2 * sigma2 * 0.5f) * //Gaussian contribution of both
			temp * // step function for 1
			temp ; // step function for the other
	}
	uk[ind].y = 0.f;

    for (int j = 0; j < dDim; j++) {
        fk[ind * dDim + j].x = 0.f;
        fk[ind * dDim + j].y = -kv[j] * uk[ind].x;
    }

}
