

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>

__global__ void d_EM_integrator(
	float* x, float* f,				// Particle positions, forces
	float delt, float noise_mag,	// Time step size, magnitude of noise
	float *L, float *Lh,			// Box length, half box length
	int *site_list,					// List of sites in group integrated
	int ns,							// Number of sites in the list
	int Dim,						// Dimensionality of the system
	hiprandState* d_states ) 
{

	int list_ind = blockIdx.x * blockDim.x + threadIdx.x;
	if (list_ind >= ns)
		return;

	int ind = site_list[list_ind];

	hiprandState l_state;

	l_state = d_states[ind];

	for (int j = 0; j < Dim; j++) {
		x[ind * Dim + j] = x[ind * Dim + j]
			+ delt * f[ind * Dim + j] +noise_mag * hiprand_normal(&l_state);

		if (x[ind * Dim + j] > L[j])
			x[ind * Dim + j] -= L[j];
		else if (x[ind * Dim + j] < 0.0)
			x[ind * Dim + j] += L[j];

	}

	d_states[ind] = l_state;
}