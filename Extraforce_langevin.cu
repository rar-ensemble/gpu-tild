
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand.h>


// Adds Langevin stochastic and friction forces to 
// allow constant T simulations with VV algorithm
__global__ void d_ExtraForce_Langevin(
    float* f,               // [ns*Dim], particle forces
    const float* v,         // [ns*Dim], particle velocities
    const float noise_mag,  // magnitude of the noise, should be sqrt(2.*gamma)
    const float gamma,      // Friction force
    const int* site_list,   // List of sites in the group
    const int ns,           // Number of sites in the list
    const int D,            // Dimensionality of the simulation
    hiprandState* d_states) {// Status of CUDA rng

    int list_ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (list_ind >= ns)
        return;

    int ind = site_list[list_ind];

    hiprandState l_state;

    l_state = d_states[ind];

    for (int j = 0; j < D; j++)
        f[ind * D + j] += -gamma * v[ind * D + j] + noise_mag * hiprand_normal(&l_state);

    d_states[ind] = l_state;

}
