#include "hip/hip_runtime.h"
#include "globals.h"
__global__ void d_copyPositions(float*, float*, int, int);


void send_3n_to_device(float** out, float *d_target) {
	int i, j;
	for (i = 0; i < ns; i++) {
		for (j = 0; j < Dim; j++) {
			h_ns_float[i * Dim + j] = out[i][j];
		}
	}
	hipMemcpy(d_target, h_ns_float, ns * Dim * sizeof(float),
		hipMemcpyHostToDevice);

	if ( using_GJF )
		d_copyPositions<<<ns_Grid, ns_Block>>>(d_xo, d_x, Dim, ns);
}


void cuda_collect_x() {
	hipMemcpy(h_ns_float, d_x, ns * Dim * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < ns; i++)
		for (int j = 0; j < Dim; j++)
			x[i][j] = h_ns_float[i * Dim + j];
}

void cuda_collect_rho() {
	hipMemcpy(all_rho, d_all_rho, ntypes * M * sizeof(float), 
		hipMemcpyDeviceToHost);
	// Copys all_rho to device_all_rho

	for (int i = 0; i < ntypes; i++) {
		for (int j = 0; j < M; j++) {
			Components[i].rho[j] = all_rho[i * M + j];
		}
	}
}



void cuda_collect_charge_density_field() {

	hipMemcpy(charge_density_field, d_charge_density_field, M * sizeof(float),
		hipMemcpyDeviceToHost);

	if (cudaReturn != hipSuccess) {
		char cherror[90];
		sprintf(cherror, "Cuda failed with error \"%s\" while cuda_collect_charge_density_field ran\n",
			hipGetErrorString(cudaReturn));
		die(cherror);
	}
}

void cuda_collect_electric_field() {

	hipMemcpy(electric_field, d_electric_field, M * Dim * sizeof(float),
		hipMemcpyDeviceToHost);

	if (cudaReturn != hipSuccess) {
		char cherror[90];
		sprintf(cherror, "Cuda failed with error \"%s\" while cuda_collect_charge_density_field ran\n",
			hipGetErrorString(cudaReturn));
		die(cherror);
	}
}


void cuda_collect_electrostatic_potential() {

	hipMemcpy(electrostatic_potential, d_electrostatic_potential, M * sizeof(float),
		hipMemcpyDeviceToHost);

	if (cudaReturn != hipSuccess) {
		char cherror[90];
		sprintf(cherror, "Cuda failed with error \"%s\" while cuda_collect_electrostatic_potential ran\n",
			hipGetErrorString(cudaReturn));
		die(cherror);
	}
}


void cuda_collect_f() {
	hipMemcpy(h_ns_float, d_f, ns * Dim * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < ns; i++)
		for (int j = 0; j < Dim; j++)
			f[i][j] = h_ns_float[i * Dim + j];
}


void send_box_params_to_device() {

	// Box geometry //
	hipMemcpy(d_L, L, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Lh, Lh, 3 * sizeof(float), hipMemcpyHostToDevice);

	hipMemcpy(d_typ, tp, ns * sizeof(int), hipMemcpyHostToDevice);

	// Bonding information //
	hipMemcpy(d_n_bonds, n_bonds, ns * sizeof(int), hipMemcpyHostToDevice);
	
	hipMemcpy(d_bond_req, bond_req, nbond_types * sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(d_bond_k, bond_k, nbond_types * sizeof(float),
		hipMemcpyHostToDevice);

	

	// Stack 2D array into 1D for transfer to device
	int* h_bond_stuff;
	h_bond_stuff = (int*)calloc(ns * MAX_BONDS, sizeof(int));
	int i, j;
	for (i = 0; i < ns; i++)
		for (j = 0; j < MAX_BONDS; j++)
			h_bond_stuff[i * MAX_BONDS + j] = bonded_to[i][j];

	hipMemcpy(d_bonded_to, h_bond_stuff, ns * MAX_BONDS * sizeof(int),
		hipMemcpyHostToDevice);

	for (i = 0; i < ns; i++)
		for (j = 0; j < MAX_BONDS; j++)
			h_bond_stuff[i * MAX_BONDS + j] = bond_type[i][j];

	hipMemcpy(d_bond_type, h_bond_stuff, ns * MAX_BONDS * sizeof(int),
		hipMemcpyHostToDevice);

	free(h_bond_stuff);


	// Copy grid information
	hipMemcpy(d_dx, dx, 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Nx, Nx, 3 * sizeof(int), hipMemcpyHostToDevice);

	// Copy masses, diffusivities
	hipMemcpy(d_mass, mass, ntypes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Diff, Diff, ntypes * sizeof(float), hipMemcpyHostToDevice);

	// Copy charges
	if (do_charges == 1) 
		hipMemcpy(d_charges, charges, ns * sizeof(float), hipMemcpyHostToDevice);
}